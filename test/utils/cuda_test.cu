
#include <stdio.h>
#include <string>
#include <scfd/utils/init_cuda.h>
#include <scfd/utils/cuda_safe_call.h>

int main(int argc, char **argv)
{
    bool    do_error = false;
    if ((argc >= 2)&&(std::string(argv[1]) == std::string("1"))) do_error = true;
    if (do_error) printf("you specified do error on purpose\n");
    try {
        scfd::utils::init_cuda(0);

        int     *p;
        if (!do_error)
            CUDA_SAFE_CALL( hipMalloc((void**)&p, sizeof(int)*512) );
        else
            CUDA_SAFE_CALL( hipMalloc((void**)&p, -100 ) );

        return 0;

    } catch (std::runtime_error &e) {
        printf("%s\n", e.what());

        return 1;
    }
}